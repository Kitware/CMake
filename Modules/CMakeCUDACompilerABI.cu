#ifndef __HIPCC__
#  error "A C or C++ compiler has been selected for CUDA"
#endif

#include <cstdio>

#include <hip/hip_runtime.h>

#include "CMakeCompilerABI.h"

int main(int argc, char* argv[])
{
  int require = 0;
  require += info_sizeof_dptr[argc];
  require += info_byte_order_big_endian[argc];
  require += info_byte_order_little_endian[argc];
#if defined(ABI_ID)
  require += info_abi[argc];
#endif
  static_cast<void>(argv);

  int count = 0;
  if (hipGetDeviceCount(&count) != hipSuccess || count == 0) {
    std::fprintf(stderr, "No CUDA devices found.\n");
    return -1;
  }

  int found = 0;
  const char* sep = "";
  for (int device = 0; device < count; ++device) {
    hipDeviceProp_t prop;
    if (hipGetDeviceProperties(&prop, device) == hipSuccess) {
      std::printf("%s%d%d", sep, prop.major, prop.minor);
      sep = ";";
      found = 1;
    }
  }

  if (!found) {
    std::fprintf(stderr, "No CUDA architecture detected from any devices.\n");
    // Convince the compiler that the non-zero return value depends
    // on the info strings so they are not optimized out.
    return require ? -1 : 1;
  }

  return 0;
}
