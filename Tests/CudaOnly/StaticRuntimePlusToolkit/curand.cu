// Comes from:
// https://docs.nvidia.com/cuda/curand/host-api-overview.html#host-api-example

#ifdef _WIN32
#  define EXPORT __declspec(dllexport)
#else
#  define EXPORT
#endif

/*
 * This program uses the host CURAND API to generate 100
 * pseudorandom floats.
 */
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <stdio.h>
#include <stdlib.h>

#define CUDA_CALL(x)                                                          \
  do {                                                                        \
    if ((x) != hipSuccess) {                                                 \
      printf("Error at %s:%d\n", __FILE__, __LINE__);                         \
      return EXIT_FAILURE;                                                    \
    }                                                                         \
  } while (0)
#define CURAND_CALL(x)                                                        \
  do {                                                                        \
    if ((x) != HIPRAND_STATUS_SUCCESS) {                                       \
      printf("Error at %s:%d\n", __FILE__, __LINE__);                         \
      return EXIT_FAILURE;                                                    \
    }                                                                         \
  } while (0)

EXPORT int curand_main()
{
  size_t n = 100;
  size_t i;
  hiprandGenerator_t gen;
  float *devData, *hostData;

  /* Allocate n floats on host */
  hostData = (float*)calloc(n, sizeof(float));

  /* Allocate n floats on device */
  CUDA_CALL(hipMalloc((void**)&devData, n * sizeof(float)));

  /* Create pseudo-random number generator */
  CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));

  /* Set seed */
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));

  /* Generate n floats on device */
  CURAND_CALL(hiprandGenerateUniform(gen, devData, n));

  /* Copy device memory to host */
  CUDA_CALL(
    hipMemcpy(hostData, devData, n * sizeof(float), hipMemcpyDeviceToHost));

  /* Cleanup */
  CURAND_CALL(hiprandDestroyGenerator(gen));
  CUDA_CALL(hipFree(devData));
  free(hostData);
  return EXIT_SUCCESS;
}
