
#include <hip/hip_runtime.h>

#ifdef _WIN32
#  define IMPORT __declspec(dllimport)
#  define EXPORT __declspec(dllexport)
#else
#  define IMPORT
#  define EXPORT
#endif

int curand_main();
int nppif_main();

EXPORT int shared_version()
{
  return curand_main() == 0 && nppif_main() == 0;
}
