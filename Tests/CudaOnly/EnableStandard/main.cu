
#include <hip/hip_runtime.h>
#ifdef _WIN32
#  define IMPORT __declspec(dllimport)
#else
#  define IMPORT
#endif

int static_cuda11_func(int);
IMPORT int shared_cuda11_func(int);

void test_functions()
{
  static_cuda11_func(int(42));
  shared_cuda11_func(int(42));
}

int main(int argc, char** argv)
{
  test_functions();
  return 0;
}
