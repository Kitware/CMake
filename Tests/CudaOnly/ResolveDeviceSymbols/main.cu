
#include <iostream>

#include "file2.h"

int file2_launch_kernel(int x);

int choose_cuda_device()
{
  int nDevices = 0;
  hipError_t err = hipGetDeviceCount(&nDevices);
  if (err != hipSuccess) {
    std::cerr << "Failed to retrieve the number of CUDA enabled devices"
              << std::endl;
    return 1;
  }
  for (int i = 0; i < nDevices; ++i) {
    hipDeviceProp_t prop;
    hipError_t err = hipGetDeviceProperties(&prop, i);
    if (err != hipSuccess) {
      std::cerr << "Could not retrieve properties from CUDA device " << i
                << std::endl;
      return 1;
    }
    std::cout << "prop.major: " << prop.major << std::endl;
    if (prop.major >= 3) {
      err = hipSetDevice(i);
      if (err != hipSuccess) {
        std::cout << "Could not select CUDA device " << i << std::endl;
      } else {
        return 0;
      }
    }
  }

  std::cout << "Could not find a CUDA enabled card supporting compute >=3.0"
            << std::endl;

  return 1;
}

int main(int argc, char** argv)
{
  int ret = choose_cuda_device();
  if (ret) {
    return 0;
  }

  file2_launch_kernel(1);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "file2_launch_kernel: kernel launch should have passed"
              << std::endl;
    return 1;
  }

  return 0;
}
