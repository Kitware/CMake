#include "hip/hip_runtime.h"

#include "file2.h"

static __global__ void file2_kernel(result_type_dynamic& r, int x)
{
  // call static_func which is a method that is defined in the
  // static library that is always out of date
  r = file2_func(x);
}

static __global__ void file2_kernel(result_type_dynamic& r, int x);

int file2_launch_kernel(int x)
{
  result_type_dynamic r;
  file2_kernel<<<1, 1>>>(r, x);
  return r.sum;
}
