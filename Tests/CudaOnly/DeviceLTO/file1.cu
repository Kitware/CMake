
#include <hip/hip_runtime.h>
#ifdef _WIN32
#  define EXPORT __declspec(dllexport)
#else
#  define EXPORT
#endif

extern __device__ int file2_func(int);
void __global__ kernel(int x)
{
  file2_func(x);
}

EXPORT int launch_kernel(int x)
{
  kernel<<<1, 1>>>(x);
  return x;
}
