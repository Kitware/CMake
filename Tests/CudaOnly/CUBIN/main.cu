#include <iostream>
#include <string>
#include <vector>

#include <hip/hip_runtime.h>

#define GENERATED_HEADER(x) GENERATED_HEADER1(x)
#define GENERATED_HEADER1(x) <x>

static std::string input_paths = { CUBIN_FILE_PATHS };

int main()
{
  const std::string delimiter = "~_~";
  input_paths += delimiter;

  size_t end = 0;
  size_t previous_end = 0;
  std::vector<std::string> actual_paths;
  while ((end = input_paths.find(delimiter, previous_end)) !=
         std::string::npos) {
    actual_paths.emplace_back(
      input_paths.substr(previous_end, end - previous_end));
    previous_end = end + 3;
  }

  hipInit(0);
  int count = 0;
  hipGetDeviceCount(&count);
  if (count == 0) {
    std::cerr << "No CUDA devices found\n";
    return 1;
  }

  hipDevice_t device;
  hipDeviceGet(&device, 0);

  hipCtx_t context;
  hipCtxCreate(&context, 0, device);

  hipModule_t module;
  for (auto p : actual_paths) {
    if (p.find(".cubin") == std::string::npos) {
      std::cout << p << " Doesn't have the .cubin suffix" << p << std::endl;
      return 1;
    }
    std::cout << "trying to load cubin: " << p << std::endl;
    hipError_t result = hipModuleLoad(&module, p.c_str());
    std::cout << "module pointer: " << module << '\n';
    if (result != hipSuccess || module == nullptr) {
      std::cerr << "Failed to load the embedded cubin with error: "
                << static_cast<unsigned int>(result) << '\n';
      return 1;
    }
  }

  return 0;
}
