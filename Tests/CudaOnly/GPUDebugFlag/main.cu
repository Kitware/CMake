#include <iostream>

#include <hip/hip_runtime.h>


static __global__ void debug_kernel(bool* has_debug)
{
// Verify using the return code if we have GPU debug flag enabled
#if defined(__CUDACC__) && defined(__CUDACC_DEBUG__)
  *has_debug = true;
#else
  *has_debug = false;
#endif
}

int choose_cuda_device()
{
  int nDevices = 0;
  hipError_t err = hipGetDeviceCount(&nDevices);
  if (err != hipSuccess) {
    std::cerr << "Failed to retrieve the number of CUDA enabled devices"
              << std::endl;
    return 1;
  }
  for (int i = 0; i < nDevices; ++i) {
    hipDeviceProp_t prop;
    hipError_t err = hipGetDeviceProperties(&prop, i);
    if (err != hipSuccess) {
      std::cerr << "Could not retrieve properties from CUDA device " << i
                << std::endl;
      return 1;
    }
    if (prop.major >= 3) {
      err = hipSetDevice(i);
      if (err != hipSuccess) {
        std::cout << "Could not select CUDA device " << i << std::endl;
      } else {
        return 0;
      }
    }
  }

  std::cout << "Could not find a CUDA enabled card supporting compute >=3.0"
            << std::endl;

  return 1;
}

int main(int argc, char** argv)
{
  bool* has_debug;
  hipError_t err = hipMallocManaged(&has_debug, sizeof(bool));
  if (err != hipSuccess) {
    std::cerr << "hipMallocManaged failed:\n"
              << "  " << hipGetErrorString(err) << std::endl;
    return 1;
  }

  debug_kernel<<<1, 1>>>(has_debug);
  err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    std::cerr << "debug_kernel: kernel launch shouldn't have failed\n"
              << "reason:\t" << hipGetErrorString(err) << std::endl;
    return 1;
  }
  if (*has_debug == false) {
    std::cerr << "debug_kernel: kernel not compiled with device debug"
              << std::endl;
    return 1;
  }
  return 0;
}
