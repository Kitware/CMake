

#include <hip/hip_runtime.h>
#include <iostream>

static __global__ void file1_kernel(int in, int* out)
{
  *out = in * in;
}

int choose_cuda_device()
{
  int nDevices = 0;
  hipError_t err = hipGetDeviceCount(&nDevices);
  if (err != hipSuccess) {
    std::cerr << "Failed to retrieve the number of CUDA enabled devices"
              << std::endl;
    return 1;
  }
  for (int i = 0; i < nDevices; ++i) {
    hipDeviceProp_t prop;
    hipError_t err = hipGetDeviceProperties(&prop, i);
    if (err != hipSuccess) {
      std::cerr << "Could not retrieve properties from CUDA device " << i
                << std::endl;
      return 1;
    }
    std::cout << "prop.major: " << prop.major << std::endl;
    if (prop.major >= 3) {
      err = hipSetDevice(i);
      if (err != hipSuccess) {
        std::cout << "Could not select CUDA device " << i << std::endl;
      } else {
        return 0;
      }
    }
  }

  std::cout << "Could not find a CUDA enabled card supporting compute >=3.0"
            << std::endl;

  return 1;
}

int file1_launch_kernel()
{
  int ret = choose_cuda_device();
  if (ret) {
    return 0;
  }

  int input = 4;

  int* output;
  hipError_t err = hipMallocManaged(&output, sizeof(int));
  hipDeviceSynchronize();
  if (err != hipSuccess) {
    return 1;
  }

  file1_kernel<<<1, 1>>>(input, output);
  hipDeviceSynchronize();
  err = hipGetLastError();
  std::cout << err << " " << hipGetErrorString(err) << std::endl;
  if (err == hipSuccess) {
    // This kernel launch should failed as the device linking never occurred
    std::cerr << "file1_kernel: kernel launch should have failed" << std::endl;
    return 1;
  }
  return 0;
}
