#include "hip/hip_runtime.h"

#include <iostream>

#include "file1.h"
#include "file2.h"

result_type __device__ file1_func(int x);
result_type_dynamic __device__ file2_func(int x);

static __global__ void file4_kernel(result_type& r, int x)
{
  // call static_func which is a method that is defined in the
  // static library that is always out of date
  r = file1_func(x);
  result_type_dynamic rd = file2_func(x);
}

EXPORT int file4_launch_kernel(int x)
{
  result_type r;
  file4_kernel<<<1, 1>>>(r, x);
  return r.sum;
}
