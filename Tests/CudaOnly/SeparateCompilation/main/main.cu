
#include <iostream>

#include "../file1.h"
#include "../file2.h"

IMPORT int file4_launch_kernel(int x);
IMPORT int file5_launch_kernel(int x);

int choose_cuda_device()
{
  int nDevices = 0;
  hipError_t err = hipGetDeviceCount(&nDevices);
  if (err != hipSuccess) {
    std::cerr << "Failed to retrieve the number of CUDA enabled devices"
              << std::endl;
    return 1;
  }
  for (int i = 0; i < nDevices; ++i) {
    hipDeviceProp_t prop;
    hipError_t err = hipGetDeviceProperties(&prop, i);
    if (err != hipSuccess) {
      std::cerr << "Could not retrieve properties from CUDA device " << i
                << std::endl;
      return 1;
    }
    if (prop.major >= 3) {
      err = hipSetDevice(i);
      if (err != hipSuccess) {
        std::cout << "Could not select CUDA device " << i << std::endl;
      } else {
        return 0;
      }
    }
  }

  std::cout << "Could not find a CUDA enabled card supporting compute >=3.0"
            << std::endl;

  return 1;
}

int main(int argc, char** argv)
{
  int ret = choose_cuda_device();
  if (ret) {
    return 0;
  }

  hipError_t err;
  file4_launch_kernel(42);
  err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "file4_launch_kernel: kernel launch failed: "
              << hipGetErrorString(err) << std::endl;
    return 1;
  }

  file5_launch_kernel(42);
  err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "file5_launch_kernel: kernel launch failed: "
              << hipGetErrorString(err) << std::endl;
    return 1;
  }

  return 0;
}
