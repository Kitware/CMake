#include <fstream>
#include <iostream>
#include <string>
#include <vector>

#include <hip/hip_runtime.h>

#define GENERATED_HEADER(x) GENERATED_HEADER1(x)
#define GENERATED_HEADER1(x) <x>

static std::string input_paths = { OPTIX_FILE_PATHS };

int main()
{
  if (input_paths == "NO_OPTIX_SUPPORT") {
    return 0;
  }

  const std::string delimiter = "~_~";
  input_paths += delimiter;

  size_t end = 0;
  size_t previous_end = 0;
  std::vector<std::string> actual_paths;
  while ((end = input_paths.find(delimiter, previous_end)) !=
         std::string::npos) {
    actual_paths.emplace_back(
      input_paths.substr(previous_end, end - previous_end));
    previous_end = end + 3;
  }

  if (actual_paths.empty()) {
    std::cerr << "Failed to parse OPTIX_FILE_PATHS" << std::endl;
    return 1;
  }

  const std::uint32_t optix_magic_value = 0x7f4e43ed;
  for (auto p : actual_paths) {
    if (p.find(".optixir") == std::string::npos) {
      std::cout << p << " Doesn't have the .optixir suffix" << p << std::endl;
      return 1;
    }
    std::ifstream input(p, std::ios::binary);
    std::uint32_t value;
    input.read(reinterpret_cast<char*>(&value), sizeof(value));
    if (value != optix_magic_value) {
      std::cerr << p << " Doesn't look like an optix-ir file" << std::endl;
      return 1;
    }
  }

  return 0;
}
