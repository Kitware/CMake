
#include <hip/hip_runtime.h>


__global__ void kernelB(float* r, float* x, float* y, float* z, int size)
{
  for (int i = threadIdx.x; i < size; i += blockDim.x) {
    r[i] = x[i] * y[i] + z[i];
  }
}
