#include "hip/hip_runtime.h"

#ifndef CUDA_PTX_COMPILATION
#  error "CUDA_PTX_COMPILATION define not provided"
#endif

__global__ void kernelB(float* r, float* x, float* y, float* z, int size)
{
  for (int i = threadIdx.x; i < size; i += blockDim.x) {
    r[i] = x[i] * y[i] + z[i];
  }
}
