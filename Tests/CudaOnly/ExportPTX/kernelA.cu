
#include <hip/hip_runtime.h>

__global__ void kernelA(float* r, float* x, float* y, float* z, int size)
{
  for (int i = threadIdx.x; i < size; i += blockDim.x) {
    r[i] = x[i] * y[i] + z[i];
  }
}
