#include <iostream>

#include <hip/hip_runtime.h>

#include "embedded_objs.h"

int main()
{
  hipInit(0);
  int count = 0;
  hipGetDeviceCount(&count);
  if (count == 0) {
    std::cerr << "No CUDA devices found\n";
    return 1;
  }

  hipDevice_t device;
  hipDeviceGet(&device, 0);

  hipCtx_t context;
  hipCtxCreate(&context, 0, device);

  hipModule_t module;
  hipError_t result = hipModuleLoadData(&module, kernels);
  std::cout << "module pointer: " << module << '\n';
  if (result != hipSuccess || module == nullptr) {
    std::cerr << "Failed to load the embedded ptx with error: "
              << static_cast<unsigned int>(result) << '\n';
    return 1;
  }
}
