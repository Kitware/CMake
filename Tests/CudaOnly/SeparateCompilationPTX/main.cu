#include <iostream>

#include <hip/hip_runtime.h>

#include "embedded_objs.h"

int main()
{
  hipInit(0);
  int count = 0;
  hipGetDeviceCount(&count);
  if (count == 0) {
    std::cerr << "No CUDA devices found\n";
    return 1;
  }

  hipDevice_t device;
  hipDeviceGet(&device, 0);

  hipCtx_t context;
  hipCtxCreate(&context, 0, device);

  hipModule_t module;
  hipModuleLoadData(&module, kernels);
  if (module == nullptr) {
    std::cerr << "Failed to load the embedded ptx" << std::endl;
    return 1;
  }
  std::cout << module << std::endl;
}
