
#include <iostream>

#include "hip/hip_runtime.h"

#ifdef _WIN32
#  define IMPORT __declspec(dllimport)
#else
#  define IMPORT
#endif

#ifndef _WIN32
IMPORT int file1_launch_kernel(int x);
#endif

IMPORT int file2_launch_kernel(int x);

int choose_cuda_device()
{
  int nDevices = 0;
  hipError_t err = hipGetDeviceCount(&nDevices);
  if (err != hipSuccess) {
    std::cerr << "Failed to retrieve the number of CUDA enabled devices"
              << std::endl;
    return 1;
  }
  for (int i = 0; i < nDevices; ++i) {
    hipDeviceProp_t prop;
    hipError_t err = hipGetDeviceProperties(&prop, i);
    if (err != hipSuccess) {
      std::cerr << "Could not retrieve properties from CUDA device " << i
                << std::endl;
      return 1;
    }
    std::cout << "prop.major: " << prop.major << std::endl;
    if (prop.major >= 3) {
      err = hipSetDevice(i);
      if (err != hipSuccess) {
        std::cout << "Could not select CUDA device " << i << std::endl;
      } else {
        return 0;
      }
    }
  }

  std::cout << "Could not find a CUDA enabled card supporting compute >=3.0"
            << std::endl;

  return 1;
}

int main(int argc, char** argv)
{
  int ret = choose_cuda_device();
  if (ret) {
    return 0;
  }

  hipError_t err;
#ifndef _WIN32
  file1_launch_kernel(1);
  err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "file1_launch_kernel: kernel launch should have passed.\n "
                 "Error message: "
              << hipGetErrorString(err) << std::endl;
    return 1;
  }
#endif

  file2_launch_kernel(1);
  err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "file2_launch_kernel: kernel launch should have passed.\n "
                 "Error message: "
              << hipGetErrorString(err) << std::endl;
    return 1;
  }

  return 0;
}
