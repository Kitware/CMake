
#include <hip/hip_runtime.h>

#ifdef _WIN32
#  define EXPORT __declspec(dllexport)
#else
#  define EXPORT
#endif

void __global__ file1_kernel(int x, int& r)
{
  r = -x;
}

EXPORT int file1_launch_kernel(int x)
{
  int r = 0;
  file1_kernel<<<1, 1>>>(x, r);
  return r;
}
