
#include <hip/hip_runtime.h>

#ifdef _WIN32
#  define EXPORT __declspec(dllexport)
#else
#  define EXPORT
#endif

void __global__ file2_kernel(int x, int& r)
{
  r = -x;
}

EXPORT int file2_launch_kernel(int x)
{
  int r = 0;
  file2_kernel<<<1, 1>>>(x, r);
  return r;
}
