
#include <hip/hip_runtime.h>

#ifdef _WIN32
#  define IMPORT __declspec(dllimport)
IMPORT int shared_version();
int static_version()
{
  return 0;
}
int mixed_version()
{
  return 0;
}
#else
int shared_version();
int static_version();
int mixed_version();
#endif

int main()
{
  return mixed_version() == 0 && shared_version() == 0 &&
    static_version() == 0;
}
