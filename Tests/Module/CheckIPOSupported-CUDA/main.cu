#include <iostream>

#include "hip/hip_runtime.h"

#ifdef _WIN32
#  define IMPORT __declspec(dllimport)
#else
#  define IMPORT
#endif

IMPORT int launch_kernel(int x);

int choose_cuda_device()
{
  int nDevices = 0;
  hipError_t err = hipGetDeviceCount(&nDevices);
  if (err != hipSuccess) {
    std::cerr << "Failed to retrieve the number of CUDA enabled devices"
              << std::endl;
    return 1;
  }
  for (int i = 0; i < nDevices; ++i) {
    hipDeviceProp_t prop;
    hipError_t err = hipGetDeviceProperties(&prop, i);
    if (err != hipSuccess) {
      std::cerr << "Could not retrieve properties from CUDA device " << i
                << std::endl;
      return 1;
    }
    std::cout << "prop.major: " << prop.major << std::endl;
    err = hipSetDevice(i);
    if (err != hipSuccess) {
      std::cout << "Could not select CUDA device " << i << std::endl;
    } else {
      return 0;
    }
  }

  std::cout << "Could not find a CUDA enabled card" << std::endl;

  return 1;
}

int main()
{
  int ret = choose_cuda_device();
  if (ret) {
    return 0;
  }

  hipError_t err;
  launch_kernel(1);
  err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "launch_kernel: kernel launch should have passed.\n "
                 "Error message: "
              << hipGetErrorString(err) << std::endl;
    return 1;
  }

  return 0;
}
