
#include <hip/hip_runtime.h>
__device__ int foo_func(int);

void __global__ bar_kernel(int x)
{
  foo_func(x);
}

int launch_kernel(int x)
{
  bar_kernel<<<1, 1>>>(x);
  return x;
}
