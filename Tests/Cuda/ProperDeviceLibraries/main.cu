#include "hip/hip_runtime.h"

#include <iostream>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#if defined(USE_THREADS_POSIX) && defined(HAVE_PTHREAD_H)

#  include <pthread.h>
static int verify_linking_to_pthread()
{
  return static_cast<int>(pthread_self());
}
#endif

// this test only makes sense for versions of CUDA that ships
// static libraries that have separable compilation device symbols
#if __CUDACC_VER_MAJOR__ <= 9
__global__ void deviceCublasSgemm(int n, float alpha, float beta,
                                  const float* d_A, const float* d_B,
                                  float* d_C)
{
  hipblasHandle_t cnpHandle;
  hipblasStatus_t status = hipblasCreate(&cnpHandle);

  if (status != HIPBLAS_STATUS_SUCCESS) {
    return;
  }

  // Call function defined in the cublas_device system static library.
  // This way we can verify that we properly pass system libraries to the
  // device link line
  status = hipblasSgemm(cnpHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha,
                       d_A, n, d_B, n, &beta, d_C, n);

  hipblasDestroy(cnpHandle);
}
#endif

int choose_cuda_device()
{
  int nDevices = 0;
  hipError_t err = hipGetDeviceCount(&nDevices);
  if (err != hipSuccess) {
    std::cerr << "Failed to retrieve the number of CUDA enabled devices"
              << std::endl;
    return 1;
  }
  for (int i = 0; i < nDevices; ++i) {
    hipDeviceProp_t prop;
    hipError_t err = hipGetDeviceProperties(&prop, i);
    if (err != hipSuccess) {
      std::cerr << "Could not retrieve properties from CUDA device " << i
                << std::endl;
      return 1;
    }

    if (prop.major > 3 || (prop.major == 3 && prop.minor >= 5)) {
      err = hipSetDevice(i);
      if (err != hipSuccess) {
        std::cout << "Could not select CUDA device " << i << std::endl;
      } else {
        return 0;
      }
    }
  }

  std::cout << "Could not find a CUDA enabled card supporting compute >=3.5"
            << std::endl;
  return 1;
}

int main(int argc, char** argv)
{
  int ret = choose_cuda_device();
  if (ret) {
    return 0;
  }

#if __CUDACC_VER_MAJOR__ <= 9
  // initial values that will make sure that the hipblasSgemm won't actually
  // do any work
  int n = 0;
  float alpha = 1;
  float beta = 1;
  float* d_A = nullptr;
  float* d_B = nullptr;
  float* d_C = nullptr;
  deviceCublasSgemm<<<1, 1>>>(n, alpha, beta, d_A, d_B, d_C);
#endif

  return 0;
}
