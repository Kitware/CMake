#include <iostream>

#include <hip/hip_runtime.h>

extern "C" int use_cuda(void)
{
  int nDevices = 0;
  hipError_t err = hipGetDeviceCount(&nDevices);
  if (err != hipSuccess) {
    std::cerr << "Failed to retrieve the number of CUDA enabled devices"
              << std::endl;
    return 1;
  }
  std::cout << "Found " << nDevices << " CUDA enabled devices" << std::endl;
  return 0;
}
