#include "hip/hip_runtime.h"

#include <iostream>

#include "file1.h"
#include "file2.h"

result_type __device__ file1_func(int x);
result_type_dynamic __device__ file2_func(int x);

static __global__ void file3_kernel(result_type* r, int x)
{
  *r = file1_func(x);
  result_type_dynamic rd = file2_func(x);
}

int file3_launch_kernel(int x)
{
  result_type* r;
  hipError_t err = hipMallocManaged(&r, sizeof(result_type));
  if (err != hipSuccess) {
    std::cerr << "file3_launch_kernel: hipMallocManaged failed: "
              << hipGetErrorString(err) << std::endl;
    return x;
  }

  file3_kernel<<<1, 1>>>(r, x);
  err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "file3_kernel [SYNC] failed: " << hipGetErrorString(err)
              << std::endl;
    return x;
  }
  err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    std::cerr << "file3_kernel [ASYNC] failed: "
              << hipGetErrorString(hipGetLastError()) << std::endl;
    return x;
  }
  int result = r->sum;
  err = hipFree(r);
  if (err != hipSuccess) {
    std::cerr << "file3_launch_kernel: hipFree failed: "
              << hipGetErrorString(err) << std::endl;
    return x;
  }

  return result;
}
