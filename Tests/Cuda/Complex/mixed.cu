#include "hip/hip_runtime.h"

#include <iostream>

#include "file1.h"
#include "file2.h"

#ifdef _WIN32
#define EXPORT __declspec(dllexport)
#define IMPORT __declspec(dllimport)
#else
#define EXPORT
#define IMPORT
#endif

result_type __device__ file1_func(int x);
result_type_dynamic __device__ file2_func(int x);

IMPORT void __host__ cuda_dynamic_lib_func();

static __global__ void mixed_kernel(result_type* r, int x)
{
  *r = file1_func(x);
  result_type_dynamic rd = file2_func(x);
}

EXPORT int mixed_launch_kernel(int x)
{
  cuda_dynamic_lib_func();

  result_type* r;
  hipError_t err = hipMallocManaged(&r, sizeof(result_type));
  if (err != hipSuccess) {
    std::cerr << "mixed_launch_kernel: hipMallocManaged failed: "
              << hipGetErrorString(err) << std::endl;
    return x;
  }

  mixed_kernel<<<1, 1>>>(r, x);
  err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "mixed_kernel [SYNC] failed: " << hipGetErrorString(err)
              << std::endl;
    return x;
  }
  err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    std::cerr << "mixed_kernel [ASYNC] failed: "
              << hipGetErrorString(hipGetLastError()) << std::endl;
    return x;
  }

  int result = r->sum;
  err = hipFree(r);
  if (err != hipSuccess) {
    std::cerr << "mixed_launch_kernel: hipFree failed: "
              << hipGetErrorString(err) << std::endl;
    return x;
  }

  return result;
}
