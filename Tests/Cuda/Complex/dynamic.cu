
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>

#ifdef _WIN32
#define EXPORT __declspec(dllexport)
#else
#define EXPORT
#endif

int dynamic_base_func(int);

EXPORT int __host__ cuda_dynamic_host_func(int x)
{
  return dynamic_base_func(x);
}

static __global__ void DetermineIfValidCudaDevice()
{
}

EXPORT int choose_cuda_device()
{
  int nDevices = 0;
  hipError_t err = hipGetDeviceCount(&nDevices);
  if (err != hipSuccess) {
    std::cerr << "Failed to retrieve the number of CUDA enabled devices"
              << std::endl;
    return 1;
  }
  for (int i = 0; i < nDevices; ++i) {
    hipDeviceProp_t prop;
    hipError_t err = hipGetDeviceProperties(&prop, i);
    if (err != hipSuccess) {
      std::cerr << "Could not retrieve properties from CUDA device " << i
                << std::endl;
      return 1;
    }
    if (prop.major >= 4) {
      err = hipSetDevice(i);
      if (err != hipSuccess) {
        std::cout << "Could not select CUDA device " << i << std::endl;
      } else {
        return 0;
      }
    }
  }

  std::cout << "Could not find a CUDA enabled card supporting compute >=3.0"
            << std::endl;

  return 1;
}

EXPORT void cuda_dynamic_lib_func()
{
  DetermineIfValidCudaDevice<<<1, 1>>>();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "DetermineIfValidCudaDevice [SYNC] failed: "
              << hipGetErrorString(err) << std::endl;
  }
  err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    std::cerr << "DetermineIfValidCudaDevice [ASYNC] failed: "
              << hipGetErrorString(hipGetLastError()) << std::endl;
  }
}
