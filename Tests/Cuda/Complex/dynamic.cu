
#include <iostream>
#include <string>

#include <hip/hip_runtime.h>

#ifdef _WIN32
#  define EXPORT __declspec(dllexport)
#else
#  define EXPORT
#endif

int dynamic_base_func(int);

EXPORT int __host__ cuda_dynamic_host_func(int x)
{
  return dynamic_base_func(x);
}

static __global__ void DetermineIfValidCudaDevice()
{
}

EXPORT int choose_cuda_device()
{
  int nDevices = 0;
  hipError_t err = hipGetDeviceCount(&nDevices);
  if (err != hipSuccess) {
    std::cerr << "Failed to retrieve the number of CUDA enabled devices"
              << std::endl;
    return 1;
  }
  for (int i = 0; i < nDevices; ++i) {
    hipDeviceProp_t prop;
    hipError_t err = hipGetDeviceProperties(&prop, i);
    if (err != hipSuccess) {
      std::cerr << "Could not retrieve properties from CUDA device " << i
                << std::endl;
      return 1;
    }
    if (prop.major >= 3) {
      err = hipSetDevice(i);
      if (err != hipSuccess) {
        std::cout << "Could not select CUDA device " << i << std::endl;
      } else {
        return 0;
      }
    }
  }

  std::cout << "Could not find a CUDA enabled card supporting compute >=3.0"
            << std::endl;

  return 1;
}

EXPORT bool cuda_dynamic_lib_func()
{
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "DetermineIfValidCudaDevice [Per Launch] failed: "
              << hipGetErrorString(err) << std::endl;
    return false;
  }
  DetermineIfValidCudaDevice<<<1, 1>>>();
  err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    std::cerr << "DetermineIfValidCudaDevice [SYNC] failed: "
              << hipGetErrorString(hipGetLastError()) << std::endl;
    return false;
  }
  return true;
}
