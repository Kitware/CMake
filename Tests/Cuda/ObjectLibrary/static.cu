
#include <hip/hip_runtime.h>

#include <iostream>

int __host__ file1_sq_func(int x)
{
  hipError_t err;
  int nDevices = 0;
  err = hipGetDeviceCount(&nDevices);
  if (err != hipSuccess) {
    std::cerr << "nDevices: " << nDevices << std::endl;
    std::cerr << "err: " << err << std::endl;
    return 1;
  }
  std::cout << "this library uses cuda code" << std::endl;
  std::cout << "you have " << nDevices << " devices that support cuda"
            << std::endl;

  return x * x;
}
