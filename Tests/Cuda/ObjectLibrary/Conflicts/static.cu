
#include <iostream>

#include <hip/hip_runtime.h>


int __host__ cu2_sq_func(int x)
{
  hipError_t err;
  int nDevices = 0;
  err = hipGetDeviceCount(&nDevices);
  if (err != hipSuccess) {
    std::cerr << "nDevices: " << nDevices << std::endl;
    std::cerr << "err: " << err << std::endl;
    return 1;
  }
  return x * x;
}
