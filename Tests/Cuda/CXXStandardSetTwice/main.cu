

#include <hip/hip_runtime.h>
#include <type_traits>

int main(int argc, char** argv)
{
  // Verify that issue #17519 Setting CXX_STANDARD breaks CUDA_STANDARD
  // selection via cxx_std_11 has been corrected
  using returnv = std::integral_constant<int, 0>;
  return returnv::value;
}
